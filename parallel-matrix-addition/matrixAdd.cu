// matrix addition c(i , j) = 1 / a(i , j) + 1 / b(i , j)
// compile by : nvcc -arch=compute_61 -code=sm_61,sm_61 -O2 -m64 -o matrixAdd matrixAdd.cu


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define BLOCKSIZE 2

// matrix in host memory
double *h_A , *h_B , *h_C ;

// matrix in device memory
double *d_A , *d_B , *d_C ;


// random initialization
void randomInit (double* matrix , int size)
{
	for(int i = 0 ; i < size ; i++) {
		for(int j = 0 ; j < size ; j++) {
			*(matrix + i * size + j) = 0.1 + rand() / (double) RAND_MAX ;
		}
	}
}

// kernel function : run in device
__global__ void matrixAdd(const double* A , const double* B , double* C , int size ) {
	int i = blockDim.y * blockIdx.y + threadIdx.y ;
	int j = blockDim.x * blockIdx.x + threadIdx.x ;

	if ( i < size && j < size  )
		*(C + i * size + j) = 1.0 / *(A + i * size + j) + 1.0 / *(B + i * size + j) ;

}


// main function : run in CPU (host)

int main(void) 
{
	int gid ;

	// Error state storing
	hipError_t err = hipSuccess ;

	printf("Enter the GPU ID : ") ;
	scanf("%d" , &gid) ;
	
	printf("%d\n" , gid) ;
	err = hipSetDevice(gid) ;
	if (err != hipSuccess) {
		printf("!!!cannot select GPU with device ID = %d\n" , gid) ;
		exit(1) ;
	}
	printf("Set GPu with device ID = %d\n" , gid) ;


	hipSetDevice(gid) ;
	printf("Matrix Addition : c(i , j) = 1 / a(i , j) + 1 / b(i , j)\n ") ;
	
	int N ;

	printf("Enter the size of the vectors : ") ;
	scanf("%d" , &N) ;
	
	printf("%d\n" ,N) ;
	
	int size = N * N * sizeof(double) ;
	
	
	// Allocate input vectors h_A , h_B , h_c
	h_A = (double*) malloc(size) ;
	h_B = (double*) malloc(size) ;
	h_C = (double*) malloc(size) ;
	
	// Initialization
	randomInit(h_A , N) ;
	randomInit(h_B , N) ;

	

	// Set the size of blocks and threads 
	dim3 threadsPerBlock(BLOCKSIZE , BLOCKSIZE);
	dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

	
	
	// create timer
	hipEvent_t start ,stop ;
	hipEventCreate(&start) ;
	hipEventCreate(&stop) ;


	// start the timer
	hipEventRecord(start , 0) ;

	// Allocate matrixs  in device memory
	hipMalloc( (void**)&d_A , size) ;
	hipMalloc( (void**)&d_B , size ) ;
	hipMalloc( (void**)&d_C , size ) ;

	// copy matrixs to device memory
	hipMemcpy( d_A , h_A , size  , hipMemcpyHostToDevice ) ;
	hipMemcpy(d_B , h_B , size  , hipMemcpyHostToDevice) ;


	// stop timer
	hipEventRecord(stop , 0) ;
	hipEventSynchronize(stop) ;


	float Intime ;
	hipEventElapsedTime(&Intime , start , stop) ;
	printf("Input time for GPU : %f (ms) \n" , Intime ) ;

	
	// start timer 
	hipEventRecord(start , 0) ;

	matrixAdd<<<blocksPerGrid , threadsPerBlock>>> (d_A , d_B , d_C , N ) ;

	err = hipGetLastError();  // check the GPU running process
	if (err != hipSuccess) {
    		printf("CUDA Kernel launch failed: %s\n", hipGetErrorString(err));
    		exit(1);
	}

	hipDeviceSynchronize() ;

	// stop timer 
	hipEventRecord(stop , 0) ;
	hipEventSynchronize(stop) ;

	float gputime ;
	hipEventElapsedTime( &gputime , start , stop ) ;
	if (gputime > 0) {
   		 printf("GPU Gflops : %f\n", 3 * N * N / (1000000 * gputime));
	}
       	else {
    		printf("GPU Gflops : N/A (gputime = 0)\n");
	}


	printf("GPU Running time : %f\n" , gputime ) ;

	
	// start timer 
	hipEventRecord(start,0);

	// copy result to host memory
	hipMemcpy(h_C , d_C , size , hipMemcpyDeviceToHost) ;

	hipFree(d_A ) ;
	hipFree(d_B) ;
	hipFree(d_C) ;
	
	// stop timer 
	hipEventRecord(stop,0);
    	hipEventSynchronize(stop);

	float Outtime ;
	hipEventElapsedTime( &Outtime , start , stop ) ;
	printf("Output time for GPU: %f (ms) \n",Outtime);


	double gpuTimeTotal = Intime + gputime + Outtime ;
	printf("Total time for GPU : %f (ms) \n" , gpuTimeTotal) ;

	

	// start running CPU 
	hipEventRecord(start , 0) ;
	double* h_D = (double*) malloc(size) ;
	for(int i = 0 ; i < N ;i++) {
		for(int j = 0 ; j < N ; j++) {
			*(h_D + i * N + j) = 1.0 / *(h_A + i * N + j) + 1.0/ *(h_B + i * N + j) ;
		}
	}

	// stop timer 
	hipEventRecord(stop , 0) ;
	hipEventSynchronize(stop) ;

	float cputime ;
	hipEventElapsedTime(&cputime , start , stop) ;
	
	float gputime_tot = Intime + Outtime + gputime ;
	printf("Processing time for CPU: %f (ms) \n",cputime);
   	printf("CPU Gflops: %f\n",3*N * N  /(1000000.0*cputime));
    	printf("Speed up of GPU = %f\n", cputime/(gputime_tot));

	
	// destroy timer 
	hipEventDestroy(start) ;
	hipEventDestroy(stop) ;


	// check result 
	printf("Check result : \n") ;
	double sum = 0 ;
	double diff ;
	for(int i = 0 ; i < N ; i++) {
		for(int j = 0 ; j < N ; j++) {
			diff = fabs( *(h_D + i * N + j) - *(h_C + i * N + j) ) ;
			sum += diff * diff ;
		}
	}

	sum = sqrt(sum) ;
	printf("norm(h_C - h_D) = %20.15e\n\n" , sum ) ;
	hipDeviceReset() ;

	free(h_A) ;
	free(h_B) ;
	free(h_C) ;
	free(h_D) ;

	return 0 ;
}

